#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include "cuda/common/book.h"

#define FRAMES	27717	// Number of Frames
#define DELTAT	10	//4// In centiseconds so time is integer
#define TAURES	2	//5// will be multplied by DELTAT
#define TRES	1	// will be multplied by DELTAT
#define TAUMAX	40	//100// will be multplied by DELTAT
#define TAUS	int(2*TAUMAX+1)	// Number of Taus
#define FRAMESHALFWINDOW	10	//25// Number of Frames, these data are averaged out
#define RESULT_TS	int(FRAMES/TRES)
#define RESULT_TAUS	int(2*int(TAUMAX/TAURES)+1)


__global__ void corr( float *aX, float *aY, float *bX, float *bY, float *c) {
    int f = blockIdx.x;
    int tau = threadIdx.x;

    if( (f < FRAMES-TAUMAX) && (f >= TAUMAX) ){
	    int f2 = f + tau - TAUMAX;
    	    c[f*TAUS+tau] = aX[f] * bX[f2] + aY[f] * bY[f2];	//GOOD
    	    //c[f*TAUS+tau] = bY[f2];	//TEST
    } else {
	    c[f*TAUS+tau]=0;
    }
}

__global__ void avrCorr( float *c, float *avrC) {
    int t = blockIdx.x;
    int tauR = threadIdx.x;
    int f = t*TRES;
    
    if( (f < FRAMES-TAUMAX) && (f >= TAUMAX) ){
	int Num=0;
	float Sum=0;
	int tau = tauR*TAURES;

	for(int f1=f-FRAMESHALFWINDOW; f1<f+FRAMESHALFWINDOW; f1++){
	    if(c[f1*TAUS+tau] != 0){
		Sum += c[f1*TAUS+tau];
		Num ++;
	    }
	}
	if(Num>0){
	    avrC[t*RESULT_TAUS+tauR] = Sum/(1.0*Num);
	} else {
	    avrC[t*RESULT_TAUS+tauR] = 0;
	}
	//    avrC[t*RESULT_TAUS+tauR] = c[f*TAUS+tau]; //TEST
    } else {
	    avrC[t*RESULT_TAUS+tauR] = 0;
    }
}	

__global__ void getmaxCorr( float *avrC, float *maxC, float *maxTau) {
    int t = blockIdx.x;
    int f = t*TRES;

    if( (f < FRAMES-TAUMAX) && (f >= TAUMAX) ){
	for (int tauR=0; tauR<RESULT_TAUS; tauR++) {
	    int tau = tauR * TAURES;
    
	    if( avrC[t*RESULT_TAUS+tauR] > maxC[t]){
		maxC[t] = avrC[t*RESULT_TAUS+tauR];
		maxTau[t] = tau;
	    }
	}
    } else {
		maxC[t] = 0;
		maxTau[t] = 0;
    }
}

int main(int argc, char **argv) {
    int t1;
    int time[FRAMES];
    float vix1, viy1, vjx1, vjy1;
    float vix[FRAMES], viy[FRAMES], vjx[FRAMES], vjy[FRAMES];
    float avrC[RESULT_TS*RESULT_TAUS];
    float maxC[RESULT_TS];
    float maxTau[RESULT_TS];
    //float dev_Cij[FRAMES*TAUS];
    float *dev_vix, *dev_viy, *dev_vjx, *dev_vjy, *dev_Cij, *dev_avrC, *dev_maxC, *dev_maxTau;
    
    // Get input parameters to read and to write
    if (argc < 4) {
	    fprintf(stderr, "Give 2 arguments for input and 2 output file names!\n");
	    exit(1);
    }
    // Input file
    FILE *ifp = fopen(argv[1], "r");
    if (ifp == NULL) {
        fprintf(stderr, "Can't open input file %s!\n", argv[1]);
        exit(1);
    }
    // Output file
    FILE *ofp = fopen(argv[2], "w");
    if (ofp == NULL) {
	fprintf(stderr, "Can't open output file %s!\n", argv[2]);
	exit(1);
    }

    // Output file2 for max Cor and Tau
    FILE *ofp2 = fopen(argv[3], "w");
    if (ofp2 == NULL) {
	fprintf(stderr, "Can't open output file %s!\n", argv[3]);
	exit(1);
    }
    //// Read data from file
    int i=0;
    float vi, vj;
    while (fscanf(ifp, "%d %f %f %f %f", &t1, &vix1, &viy1, &vjx1, &vjy1) != EOF) {
	//fprintf(ofp, "%d %.4f\n", t1, vix1);	//TEST
	time[i]=t1;
	vi=sqrt(vix1*vix1+viy1*viy1);
	if(vi>0){
	    vix[i]=vix1/vi;
	    viy[i]=viy1/vi;
	} else {
	    vix[i]=0;
	    viy[i]=0;
	}
	vj=sqrt(vjx1*vjx1+vjy1*vjy1);
	if(vj>0){
	    vjx[i]=vjx1/vj;
	    vjy[i]=vjy1/vj;
	} else {
	    vjx[i]=0;
	    vjy[i]=0;
	}
	i++;
    }
    //// Close input file
    fclose(ifp);
    
    //// Initialize output
    for (int t=0; t<RESULT_TS; t++) {
    	maxC[t]=0;
    	maxTau[t]=0;
	for (int tau=0; tau<RESULT_TAUS; tau++) {
    	    avrC[t*RESULT_TAUS+tau]=0;
	}
    }

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_vix, FRAMES * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_viy, FRAMES * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_vjx, FRAMES * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_vjy, FRAMES * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_Cij, FRAMES*TAUS * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_avrC, RESULT_TS*RESULT_TAUS * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_maxC, RESULT_TS * sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_maxTau, RESULT_TS * sizeof(float) ) );

    // copy the v arrays to the GPU
    HANDLE_ERROR( hipMemcpy( dev_vix, vix, FRAMES * sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_viy, viy, FRAMES * sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_vjx, vjx, FRAMES * sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_vjy, vjy, FRAMES * sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_maxC, maxC, RESULT_TS * sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_maxTau, maxTau, RESULT_TS * sizeof(float), hipMemcpyHostToDevice ) );

    corr<<<FRAMES,TAUS>>>(dev_vix, dev_viy, dev_vjx, dev_vjy, dev_Cij);
    avrCorr<<<RESULT_TS,RESULT_TAUS>>>(dev_Cij, dev_avrC);
    getmaxCorr<<<RESULT_TS,1>>>(dev_avrC, dev_maxC, dev_maxTau);

    //// copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( avrC, dev_avrC, RESULT_TS*RESULT_TAUS * sizeof(float), hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipMemcpy( maxC, dev_maxC, RESULT_TS * sizeof(float), hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipMemcpy( maxTau, dev_maxTau, RESULT_TS * sizeof(float), hipMemcpyDeviceToHost ) );
    

    fprintf(ofp, "#%s\n", argv[1] );		fprintf(ofp2, "#%s\n", argv[1] );		
    fprintf(ofp, "#FRAMES: %d\n", FRAMES );     fprintf(ofp2, "#FRAMES: %d\n", FRAMES );
    fprintf(ofp, "#DELTAT: %d\n", DELTAT );     fprintf(ofp2, "#DELTAT: %d\n", DELTAT );
    fprintf(ofp, "#TAURES: %d\n", TAURES );     fprintf(ofp2, "#TAURES: %d\n", TAURES );
    fprintf(ofp, "#TRES: %d\n",   TRES );       fprintf(ofp2, "#TRES: %d\n",   TRES );
    fprintf(ofp, "#TAUMAX: %d\n", TAUMAX );     fprintf(ofp2, "#TAUMAX: %d\n", TAUMAX );
    fprintf(ofp, "#time\t");		        fprintf(ofp2, "#time\tmaxCor\tmaxTau\n");
    for (int tauR=0; tauR<RESULT_TAUS; tauR++) {
	int tau = tauR * TAURES;
    	fprintf(ofp, "%d\t", (tau-TAUMAX)*DELTAT);
    }
    fprintf(ofp, "\n");
    // display the results
    int f=0;
    for (int t=0; t<RESULT_TS; t++) {
	f = t*TRES;
    	fprintf(ofp, "%d\t", time[f]);
	for (int tauR=0; tauR<RESULT_TAUS; tauR++) {
    	    fprintf(ofp, "%.4f\t", avrC[t*RESULT_TAUS+tauR] );
	}
	fprintf(ofp, "\n");
    	fprintf(ofp2, "%d\t%.4f\t%.4f\n", time[f], maxC[t], (maxTau[t]-TAUMAX)*DELTAT );
    }


    // free the memory allocated on the GPU
    HANDLE_ERROR( hipFree( dev_vix ) );
    HANDLE_ERROR( hipFree( dev_viy ) );
    HANDLE_ERROR( hipFree( dev_vjx ) );
    HANDLE_ERROR( hipFree( dev_vjy ) );
    HANDLE_ERROR( hipFree( dev_Cij ) );
    HANDLE_ERROR( hipFree( dev_avrC ) );
    HANDLE_ERROR( hipFree( dev_maxC ) );
    HANDLE_ERROR( hipFree( dev_maxTau ) );

    //// Close output file
    fclose(ofp);
    fclose(ofp2);

    return 0;
}